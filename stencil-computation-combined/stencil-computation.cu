#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <omp.h>
#include "hip/hip_runtime.h"


//Prototypes
static float* executeSerialImplementation(void);
static void computeSerialGrid(float* read, float* write);
static float* executeOpenMPImplementation(void);
static void computeOpenMPGrid(float* read, float* write);
static float* executeCUDAImplementation(void);
static __global__ void computeCUDAGrid(float* read, float* write);
static void fillGrid(float* grid);
static void swapGrids(float** read, float** write);
static void verify(float* a, float* b);

//Control variables
const int SIZE = 30000;									//Length and width of inner grid in threads
const int DIM = SIZE + 2;								//Length and width of the entire grid in threads
const int GRID_SIZE = 1500;								//Length and width of inner grid in blocks
const int BLOCK_SIZE = 20;								//Length and width of block in threads
const size_t MEM_SIZE = sizeof(float) * DIM * DIM;		//Amount of memory used by a single grid
const int TIME_STEPS = 10;								//Number of time steps to perform
const int PINNED = 0;									//Controls using pinned and unpinned CUDA host memory
const int NUM_THREADS = 16;								//Controls the number of threads used by OpenMP

//Performs the execution of each implementation and the comparison of each result
int main(void) {
	//Executes each implementation and gets each result
	//float* serial = executeSerialImplementation();
	//float* omp = executeOpenMPImplementation();
	executeCUDAImplementation();

	//Compares the results of each implementation
	//Transitively, if a == b and b == c, we already know a == c
	//verify(serial, omp);
	//verify(omp, cuda);

	//Frees the write grids after verification
	//free(serial);
	//free(omp);
	//if (PINNED) {
	//	cudaFreeHost(cuda);
	//}
	//else {
	//	free(cuda);
	//}

	return 0;
}

//Executes the serial implementation of the stencil computation
static float* executeSerialImplementation(void) {
	//Allocate memory for the read and write grid
	float* read = (float*) malloc(MEM_SIZE);
	float* write = (float*) malloc(MEM_SIZE);
	assert(read != NULL && write != NULL);

	//Fill in the read grid
	fillGrid(read);

	//These variables keep track of the amount of time it takes for the grid computation to take
	double serialComputeTime = 0;
	double start;
	double end;

	//Compute the write grid TIME_STEPS times
	//Finally, swap the grids 1 last time to get result into write grid
	for (int i = 0; i < TIME_STEPS; i++) {
		start = omp_get_wtime();
		computeSerialGrid(read, write);
		end = omp_get_wtime();

		serialComputeTime += end - start;
		swapGrids(&read, &write);
	}
	swapGrids(&read, &write);

	//Prints out the timing information
	printf("Total serial compute time: %.5lf seconds\n", serialComputeTime);

	//Free all but the write grid
	free(read);
	
	return write;
}

//Computes the write grid for the serial implementation
static void computeSerialGrid(float* read, float* write) {
	//The result of a cell is the sum of its neighbors
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
		}
	}
}

//Executes the OpenMP implementation of the stencil computation
static float* executeOpenMPImplementation(void) {
	//Disables dynamic teams to force the max number of threads to always be used
	omp_set_dynamic(0);

	//Sets the max number of threads to use for all parallel operations
	omp_set_num_threads(NUM_THREADS);

	//Allocate memory for the read and write grids
	float* read = (float*) malloc(MEM_SIZE);
	float* write = (float*) malloc(MEM_SIZE);
	assert(read != NULL && write != NULL);

	//Fill in the read grid
	fillGrid(read);

	//These variables keep track of the amount of time it takes for the grid computation to take
	double ompComputeTime = 0;
	double start;
	double end;

	//Compute the write grid TIME_STEPS times
	//Finally, swap the grids 1 last time to get result into write grid
	for (int i = 0; i < TIME_STEPS; i++) {
		start = omp_get_wtime();
		computeOpenMPGrid(read, write);
		end = omp_get_wtime();

		ompComputeTime += end - start;
		swapGrids(&read, &write);
	}
	swapGrids(&read, &write);

	//Prints out the timing information
	printf("Total OpenMP compute time: %.5lf seconds\n", ompComputeTime);

	//Frees all but the write grid
	free(read);
	
	return write;
}

//Computes the write grid of the OpenMP implementation
static void computeOpenMPGrid(float* read, float* write) {
	//The result of a cell is the sum of its neighbors
	#pragma omp parallel for collapse(2)
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
		}
	}
}

//Performs the CUDA implementation of the stencil computation
static float* executeCUDAImplementation(void) {
	//The total number of threads must match the amount specified by the GRID_SIZE and BLOCK_SIZE
	assert(GRID_SIZE * BLOCK_SIZE == SIZE);

	//Allocates space on the host for the grids in pinned or paged memory, depending on PINNED
	//Using	pinned memory is much faster, only being limited by the speed of the PCI-E bus
	//Pinned memory, however, takes longer to allocate
	float* hostRead;
	float* hostWrite;
	if (PINNED) {
		assert(hipHostMalloc((void**) &hostRead, MEM_SIZE, hipHostMallocDefault) == hipSuccess);
		assert(hipHostMalloc((void**) &hostWrite, MEM_SIZE, hipHostMallocDefault) == hipSuccess);
	}
	else {
		double start = omp_get_wtime();
		hostRead = (float*) malloc(MEM_SIZE);
		hostWrite = (float*) malloc(MEM_SIZE);
		double end = omp_get_wtime();
		printf("Total CUDA host allocation time: %.5lf seconds\n", end - start);
		assert(hostRead != NULL && hostWrite != NULL);
	}

	//Fills in the read grid serially
	fillGrid(hostRead);

	//Allocates space on the device for the grids, and copies over the input grid
	float* deviceRead;
	float* deviceWrite;
	assert(hipMalloc(&deviceRead, MEM_SIZE) == hipSuccess);
	assert(hipMalloc(&deviceWrite, MEM_SIZE) == hipSuccess);
	assert(hipMemcpy(deviceRead, hostRead, MEM_SIZE, hipMemcpyHostToDevice) == hipSuccess);

	//Calls the computeGrid kernel TIME_STEPS times, swapping input and output each time
	//Finally, swap the grids 1 last time to get the result into the deviceWrite grid
	dim3 gridDimensions(GRID_SIZE, GRID_SIZE);
	dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
	for (int i = 0; i < TIME_STEPS; i++) {
		computeCUDAGrid<<<gridDimensions, blockDimensions>>>(deviceRead, deviceWrite);
		hipDeviceSynchronize();
		swapGrids(&deviceRead, &deviceWrite);
	}
	swapGrids(&deviceRead, &deviceWrite);

	//Copies over the result (now in deviceWrite) from device to host
	assert(hipMemcpy(hostWrite, deviceWrite, MEM_SIZE, hipMemcpyDeviceToHost) == hipSuccess);

	//Frees all but the host write grid
	if (PINNED) {
		hipHostFree(hostRead);
		hipHostFree(hostWrite);
	}
	else {
		double start = omp_get_wtime();
		free(hostRead);
		free(hostWrite);
		double end = omp_get_wtime();
		printf("Total CUDA host free time: %.5lf seconds\n", end - start);
	}
	hipFree(deviceRead);
	hipFree(deviceWrite);

	return NULL;
}

//Performs a parallelized stencil computation using CUDA cores
static __global__ void computeCUDAGrid(float* read, float* write) {
	//Retrieve the thread's position in the grid
	//The position is offset by 1 in the x and y directions to remove boundary checks
	int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int y = blockDim.y * blockIdx.y + threadIdx.y + 1;

	//Writes the sum of the neighbors to the cell
	write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
}

//Fills in a grid serially with sample data
static void fillGrid(float* grid) {
	//Fills in the leftmost and rightmost columns
	for (int y = 0; y < DIM; y++) {
		grid[DIM * y] = grid[DIM * y + DIM - 1] = 0;
	}

	//Fills in the top and bottom rows
	for (int x = 0; x < DIM; x++) {
		grid[x] = grid[DIM * (DIM - 1) + x] = 0;
	}

	//Fills in each spot of the inner grid with 1.1
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			grid[DIM * y + x] = 1.1;
		}
	}
}

//Swaps the pointers of two grids
static void swapGrids(float** read, float** write) {
	float* temp = *read;
	*read = *write;
	*write = temp;
}

//Verifies that two grids are equal to each other
static void verify(float* a, float* b) {
	assert(memcmp(a, b, MEM_SIZE) == 0);
}