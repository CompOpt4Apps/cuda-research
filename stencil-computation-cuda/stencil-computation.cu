#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"


#define SIZE 16384				//Length and width of inner grid in threads
#define DIM (SIZE + 2)			//Length and width of the entire grid in threads
#define GRID_SIZE 1024 			//Length and width of inner grid in blocks
#define BLOCK_SIZE 16 			//Length and width of block in threads
#define MEM_SIZE (sizeof(float) * DIM * DIM)
#define TIME_STEPS 50

void fillGrid(float* grid);
__global__ void computeGrid(float* read, float* write);
void swapGrids(float** read, float** write);
void printGrid(float* grid, const char* name);

int main(void) {
	//The total number of threads must match the amount specified by the GRID_SIZE and BLOCK_SIZE
	assert(GRID_SIZE * BLOCK_SIZE == SIZE);

	//Allocates space on the host for the grids in pinned memory
	//Using	pinned memory is much faster, only being limited by the speed of the PCI-E bus
	float* hostRead;
	float* hostWrite;
	assert(hipHostMalloc((void**) &hostRead, MEM_SIZE, hipHostMallocDefault) == hipSuccess);
	assert(hipHostMalloc((void**) &hostWrite, MEM_SIZE, hipHostMallocDefault) == hipSuccess);

	//Fills in the read grid serially
	fillGrid(hostRead);

	//Allocates space on the device for the grids, and copies over the input grid
	float* deviceRead;
	float* deviceWrite;
	assert(hipMalloc(&deviceRead, MEM_SIZE) == hipSuccess);
	assert(hipMalloc(&deviceWrite, MEM_SIZE) == hipSuccess);
	assert(hipMemcpy(deviceRead, hostRead, MEM_SIZE, hipMemcpyHostToDevice) == hipSuccess);

	//Calls the computeGrid kernel TIME_STEPS times, swapping input and output each time
	//Finally, swap the grids 1 last time to get the result into the deviceWrite grid
	dim3 gridDimensions(GRID_SIZE, GRID_SIZE);
	dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
	for (int i = 0; i < TIME_STEPS; i++) {
		computeGrid<<<gridDimensions, blockDimensions>>>(deviceRead, deviceWrite);
		hipDeviceSynchronize();
		swapGrids(&deviceRead, &deviceWrite);
	}
	swapGrids(&deviceRead, &deviceWrite);

	//Copies over the result (now in deviceRead) from device to host
	assert(hipMemcpy(hostWrite, deviceWrite, MEM_SIZE, hipMemcpyDeviceToHost) == hipSuccess);
	
	//Print out state of the grid
	//printGrid(hostWrite, "Result");

	//Frees both the host and device grids
	hipHostFree(hostRead);
	hipHostFree(hostWrite);
	hipFree(deviceRead);
	hipFree(deviceWrite);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
void fillGrid(float* grid) {
	//Fills in the leftmost and rightmost columns
	for (int y = 0; y < DIM; y++) {
		grid[DIM * y] = grid[DIM * y + DIM - 1] = 0;
	}

	//Fills in the top and bottom rows
	for (int x = 0; x < DIM; x++) {
		grid[x] = grid[DIM * (DIM - 1) + x] = 0;
	}

	//Fills in each spot of the inner grid with 1.1
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			grid[DIM * y + x] = 1.1;
		}
	}
}

//Performs a parallelized stencil computation
__global__ void computeGrid(float* read, float* write) {
	//Retrieve the thread's position in the grid
	//The position is offset by 1 in the x and y directions to remove boundary checks
	int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int y = blockDim.y * blockIdx.y + threadIdx.y + 1;

	//Writes the sum of the neighbors to the cell
	write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
}

//Swaps the pointers of two grids
void swapGrids(float** read, float** write) {
	float* temp = *read;
	*read = *write;
	*write = temp;
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(float* grid, const char* name) {
	//Prints the name of the grid
	printf("<<< %s >>>\n\n", name);

	//Prints the inner grid
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			printf("%-25.3f", grid[DIM * y + x]);
		}

		printf("\n");
	}
}