#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include "hip/hip_runtime.h"

#include <omp.h>

#define SIZE 30000				//Length and width of inner grid in threads
#define DIM (SIZE + 2)			//Length and width of the entire grid in threads
#define GRID_SIZE 1500 			//Length and width of inner grid in blocks
#define BLOCK_SIZE 20 			//Length and width of block in threads
#define MEM_SIZE (sizeof(float) * DIM * DIM)
#define TIME_STEPS 1
#define PINNED 0

void fillGrid(float* grid);
__global__ void computeGrid(float* read, float* write);
void swapGrids(float** read, float** write);
void printGrid(float* grid, const char* name);

void computeGrid2(float* read, float* write) {
	//The result of a cell is the sum of its neighbors
	#pragma omp parallel for collapse(2)
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
		}
	}
}

float* getOMPResult() {
	//Disables dynamic teams to force the max number of threads to always be used
	omp_set_dynamic(0);

	//Sets the max number of threads to use for all parallel operations
	omp_set_num_threads(8);

	//Allocate memory for the read and write grid
	float* read = (float*) malloc(MEM_SIZE);
	float* write = (float*) malloc(MEM_SIZE);
	assert(read != NULL && write != NULL);

	//Fill in the read grid
	fillGrid(read);
	//Compute the write grid TIME_STEPS times
	//Finally, swap the grids 1 last time to get result into write grid
	for (int i = 0; i < TIME_STEPS; i++) {
		computeGrid2(read, write);
		swapGrids(&read, &write);
	}
	swapGrids(&read, &write);

	return write;
}

int main(void) {
	//The total number of threads must match the amount specified by the GRID_SIZE and BLOCK_SIZE
	assert(GRID_SIZE * BLOCK_SIZE == SIZE);

	//Allocates space on the host for the grids in pinned or paged memory, depending on PINNED
	//Using	pinned memory is much faster, only being limited by the speed of the PCI-E bus
	//Pinned memory, however, takes longer to allocate
	float* hostRead;
	float* hostWrite;
	if (PINNED) {
		assert(hipHostMalloc((void**) &hostRead, MEM_SIZE, hipHostMallocDefault) == hipSuccess);
		assert(hipHostMalloc((void**) &hostWrite, MEM_SIZE, hipHostMallocDefault) == hipSuccess);
	}
	else {
		hostRead = (float*) malloc(MEM_SIZE);
		hostWrite = (float*) malloc(MEM_SIZE);
		assert(hostRead != NULL && hostWrite != NULL);
	}

	//Fills in the read grid serially
	fillGrid(hostRead);

	//Allocates space on the device for the grids, and copies over the input grid
	float* deviceRead;
	float* deviceWrite;
	assert(hipMalloc(&deviceRead, MEM_SIZE) == hipSuccess);
	assert(hipMalloc(&deviceWrite, MEM_SIZE) == hipSuccess);
	assert(hipMemcpy(deviceRead, hostRead, MEM_SIZE, hipMemcpyHostToDevice) == hipSuccess);

	//Calls the computeGrid kernel TIME_STEPS times, swapping input and output each time
	//Finally, swap the grids 1 last time to get the result into the deviceWrite grid
	dim3 gridDimensions(GRID_SIZE, GRID_SIZE);
	dim3 blockDimensions(BLOCK_SIZE, BLOCK_SIZE);
	for (int i = 0; i < TIME_STEPS; i++) {
		computeGrid<<<gridDimensions, blockDimensions>>>(deviceRead, deviceWrite);
		hipDeviceSynchronize();
		swapGrids(&deviceRead, &deviceWrite);
	}
	swapGrids(&deviceRead, &deviceWrite);

	//Copies over the result (now in deviceRead) from device to host
	assert(hipMemcpy(hostWrite, deviceWrite, MEM_SIZE, hipMemcpyDeviceToHost) == hipSuccess);
	assert(memcmp(hostWrite, getOMPResult(), MEM_SIZE) == 0);

	//Print out state of the grid
	//printGrid(hostWrite, "Result");

	//Frees both the host and device grids
	if (PINNED) {
		hipHostFree(hostRead);
		hipHostFree(hostWrite);
	}
	else {
		free(hostRead);
		free(hostWrite);
	}
	hipFree(deviceRead);
	hipFree(deviceWrite);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
void fillGrid(float* grid) {
	//Fills in the leftmost and rightmost columns
	for (int y = 0; y < DIM; y++) {
		grid[DIM * y] = grid[DIM * y + DIM - 1] = 0;
	}

	//Fills in the top and bottom rows
	for (int x = 0; x < DIM; x++) {
		grid[x] = grid[DIM * (DIM - 1) + x] = 0;
	}

	//Fills in each spot of the inner grid with 1.1
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			grid[DIM * y + x] = 1.1;
		}
	}
}

//Performs a parallelized stencil computation
__global__ void computeGrid(float* read, float* write) {
	//Retrieve the thread's position in the grid
	//The position is offset by 1 in the x and y directions to remove boundary checks
	int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int y = blockDim.y * blockIdx.y + threadIdx.y + 1;

	//Writes the sum of the neighbors to the cell
	write[DIM * y + x] = read[DIM * (y - 1) + x] + read[DIM * (y + 1) + x] + read[DIM * y + x - 1] + read[DIM * y + x + 1];
}

//Swaps the pointers of two grids
void swapGrids(float** read, float** write) {
	float* temp = *read;
	*read = *write;
	*write = temp;
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(float* grid, const char* name) {
	//Prints the name of the grid
	printf("<<< %s >>>\n\n", name);

	//Prints the inner grid
	for (int y = 1; y < DIM - 1; y++) {
		for (int x = 1; x < DIM - 1; x++) {
			printf("%-25.3f", grid[DIM * y + x]);
		}

		printf("\n");
	}
}