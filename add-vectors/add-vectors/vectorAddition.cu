#include <stdio.h>
#include "hip/hip_runtime.h"


#define SIZE 100

//Prototypes
__global__ void addVectors(int* A, int* B, int* result);

int main() {
	//Allocates memory for the vectors
	int* A;
	int* B;
	int* result;
	hipMallocManaged(&A, SIZE * sizeof(int));
	hipMallocManaged(&B, SIZE * sizeof(int));
	hipMallocManaged(&result, SIZE * sizeof(int));

	//Initializes the vectors
	for (int i = 0; i < SIZE; i++) {
		*(A + i) = i;
		*(B + i) = i * i;
	}

	//Adds the vectors together
	addVectors<<<1, SIZE>>>(A, B, result);
	hipDeviceSynchronize();

	//Print the result
	printf("Resulting vector: [");
	for (int i = 0; i < SIZE; i++) {
		printf("%s%d", i == 0 ? "" : ", ", *(result + i));
	}
	printf("]\n");

	//Frees the vectors
	hipFree(A);
	hipFree(B);
	hipFree(result);

	return 0;
}

//Adds two vectors together
__global__ void addVectors(int* A, int* B, int* result) {
	*(result + threadIdx.x) = *(A + threadIdx.x) + *(B + threadIdx.x);
}