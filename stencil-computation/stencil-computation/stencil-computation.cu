#include <stdio.h>
#include "hip/hip_runtime.h"


#define THREAD_DIM 20							//Square dimensions of the ENTIRE grid, including inner grid, whose dimension is 2 less
#define GRID_DIM 4								//Square dimensions of the grid in blocks
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)		//Square dimensions of a single block, which only works if THREAD_DIM % GRID_DIM == 0

//Prototypes
__global__ void fillGrid(int* grid);
__global__ void computeGrid(int* grid, int* result);
void printGrid(int* grid, char* name);

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocate memory for the grid (1D array)
	int* grid;
	hipMallocManaged(&grid, THREAD_DIM * THREAD_DIM * sizeof(int));

	//Fill in the grid
	fillGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid);
	hipDeviceSynchronize();

	//Print out state of the grid
	printGrid(grid, "filled");
	printf("\n\n");

	//Allocate memory for the resulting grid
	int* result;
	hipMallocManaged(&result, THREAD_DIM * THREAD_DIM * sizeof(int));

	//Compute the resulting grid
	computeGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid, result);
	hipDeviceSynchronize();

	//Print out the resulting grid
	printGrid(result, "result");

	//Frees the memory used by the grids
	hipFree(grid);
	hipFree(result);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
__global__ void fillGrid(int* grid) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If the grid point is along the edges, fill in 0
	//Otherwise, fill in with value dependent on x and y values
	*(grid + THREAD_DIM * y + x) = x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1 ? 0 : x * y;
}

//Computes the result grid by adding all neighbors
__global__ void computeGrid(int* grid, int* result) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If along the edges of the grid, don't add neighbors
	if (x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1) {
		*(result + THREAD_DIM * y + x) = 0;
		return;
	}

	//Gets each of the neighbor's values
	int north = y == 0 ? 0 : *(grid + THREAD_DIM * (y - 1) + x);
	int south = y == THREAD_DIM - 1 ? 0 : *(grid + THREAD_DIM * (y + 1) + x);
	int west = x == 0 ? 0 : *(grid + THREAD_DIM * y + x - 1);
	int east = x == THREAD_DIM - 1 ? 0 : *(grid + THREAD_DIM * y + x + 1);

	//Put the sum of the neighbors in the result
	*(result + THREAD_DIM * y + x) = north + south + west + east;
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid, char* name) {
	printf("<<< State of the grid \"%s\" >>>\n\n", name);

	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			printf("%-10d", *(grid + THREAD_DIM * y + x));
		}

		printf("\n");
	}
}