#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#define THREAD_DIM 20							//Square dimensions of the ENTIRE grid, including inner grid, whose dimension is 2 less
#define GRID_DIM 4								//Square dimensions of the grid in blocks
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)		//Square dimensions of a single block, which only works if THREAD_DIM % GRID_DIM == 0

//Prototypes
__global__ void fillGrid(int* grid);
__global__ void computeGrid(int* grid, int* result);
void printGrid(int* grid);

static int createOutput = 0;

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocate memory for the grid (1D array) and the result grid
	int* grid;
	int* result;
	hipMallocManaged(&grid, THREAD_DIM * THREAD_DIM * sizeof(int));
	hipMallocManaged(&result, THREAD_DIM * THREAD_DIM * sizeof(int));

	//Fill in the grid and compute the result
	fillGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM - 1, BLOCK_DIM - 1)>>>(grid);
	hipDeviceSynchronize();
	computeGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM - 1, BLOCK_DIM - 1)>>>(grid, result);
	hipDeviceSynchronize();

	if (createOutput) {
		//Print out state of the filled and result grids
		printGrid(grid, "Filled");
		printf("\n");
		printGrid(result, "Result");
	}

	//Frees the memory used by the grids
	hipFree(grid);
	hipFree(result);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
__global__ void fillGrid(int* grid) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If the grid point is along the edges, fill in 0
	//Otherwise, fill in with value dependent on x and y values
	*(grid + THREAD_DIM * y + x) = x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1 ? 0 : x * y;
}

//Computes the result grid by adding all neighbors
__global__ void computeGrid(int* grid, int* result) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If along the edges of the grid, don't add neighbors
	if (x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1) {
		return;
	}

	//Put the sum of the neighbors in the result
	*(result + THREAD_DIM * y + x) = *(grid + THREAD_DIM * (y - 1) + x) + *(grid + THREAD_DIM * (y + 1) + x) + *(grid + THREAD_DIM * y + x - 1) + *(grid + THREAD_DIM * y + x + 1);
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid, char* name) {
	printf("<<< %s >>>\n\n", name);

	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			printf("%-10d", *(grid + THREAD_DIM * y + x));
		}

		printf("\n");
	}
}