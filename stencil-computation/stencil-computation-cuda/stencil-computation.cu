/*
 * Task List
 *
 * 1) Optimize CUDA implementation.
 * 2) Make a serial CPU implementation.
 * 3) Make an OpenMP parallel implementation (in same file).
 * 4) Do a speedup analysis of parallel vs serial implementations.
 */

#include <stdio.h>
#include "hip/hip_runtime.h"


#define THREAD_DIM 20							//Square dimensions of the ENTIRE grid, including inner grid, whose dimension is 2 less
#define GRID_DIM 4								//Square dimensions of the grid in blocks
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)		//Square dimensions of a single block, which only works if THREAD_DIM % GRID_DIM == 0

//Prototypes
__global__ void fillGrid(int* grid);
__global__ void computeGrid(int* grid, int* result);
void printGrid(int* grid);

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocate memory for the grid (1D array)
	int* grid;
	hipMallocManaged(&grid, THREAD_DIM * THREAD_DIM * sizeof(int));

	//Fill in the grid, record time elapsed
	fillGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid);
	hipDeviceSynchronize();

	//Print out state of the grid
	printf("<<< Filled >>>\n\n");
	printGrid(grid);
	printf("\n");

	//Allocate memory for the resulting grid
	int* result;
	hipMallocManaged(&result, THREAD_DIM * THREAD_DIM * sizeof(int));

	//Compute the resulting grid, record time elapsed
	computeGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid, result);
	hipDeviceSynchronize();

	//Print out the resulting grid and timing results
	printf("<<< Result >>>\n\n");
	printGrid(result);

	//Frees the memory used by the grids
	hipFree(grid);
	hipFree(result);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
__global__ void fillGrid(int* grid) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If the grid point is along the edges, fill in 0
	//Otherwise, fill in with value dependent on x and y values
	*(grid + THREAD_DIM * y + x) = x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1 ? 0 : x * y;
}

//Computes the result grid by adding all neighbors
__global__ void computeGrid(int* grid, int* result) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If along the edges of the grid, don't add neighbors
	if (x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1) {
		return;
	}

	//Put the sum of the neighbors in the result
	*(result + THREAD_DIM * y + x) = *(grid + THREAD_DIM * (y - 1) + x) + *(grid + THREAD_DIM * (y + 1) + x) + *(grid + THREAD_DIM * y + x - 1) + *(grid + THREAD_DIM * y + x + 1);
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid) {
	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			printf("%-10d", *(grid + THREAD_DIM * y + x));
		}

		printf("\n");
	}
}