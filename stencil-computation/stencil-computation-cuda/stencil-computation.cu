#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"


#define THREAD_DIM 12
#define GRID_DIM 2
#define BLOCK_DIM ((THREAD_DIM - 2) / GRID_DIM)
#define MEM_SIZE (sizeof(int) * THREAD_DIM * THREAD_DIM)
#define TIME_STEPS 1000000

void fillGrid(int* grid);
__global__ void computeGrid(int* read, int* write);
void swapGrids(int** read, int** write);
void printGrid(int* grid, int skip, const char* name);

int main() {
	//Checks for a valid grid dimension
	if ((THREAD_DIM - 2) % GRID_DIM != 0) {
		printf("Error: bad grid/thread dimensions.\n");
		return 1;
	}

	//Allocates space on the host for the grids in pinned memory
	//Using	memory is much faster, only being limited by the speed of the PCI-E bus
	int* hostRead;
	int* hostWrite;
	hipHostMalloc((void**) &hostRead, MEM_SIZE, hipHostMallocDefault);
	hipHostMalloc((void**) &hostWrite, MEM_SIZE, hipHostMallocDefault);

	//Fills in the read grid serially
	fillGrid(hostRead);

	//Allocates space on the device for the grids, and copies over the input grid
	int* deviceRead;
	int* deviceWrite;
	hipMalloc(&deviceRead, MEM_SIZE);
	hipMalloc(&deviceWrite, MEM_SIZE);
	hipMemcpy(deviceRead, hostRead, MEM_SIZE, hipMemcpyHostToDevice);

	//Calls the computeGrid kernel TIME_STEPS times, swapping input and output each time
	dim3 gridDimension(GRID_DIM, GRID_DIM);
	dim3 blockDimension(BLOCK_DIM, BLOCK_DIM);
	for (int i = 0; i < TIME_STEPS; i++) {
		computeGrid<<<gridDimension, blockDimension>>>(deviceRead, deviceWrite);
		hipDeviceSynchronize();
		swapGrids(&deviceRead, &deviceWrite);
	}

	//Copies over the result (now in deviceRead) from device to host
	hipMemcpy(hostWrite, deviceRead, MEM_SIZE, hipMemcpyDeviceToHost);
	printGrid(hostWrite, 1, "Result");

	//Frees both the host and device grids
	hipHostFree(hostRead);
	hipHostFree(hostWrite);
	hipFree(deviceRead);
	hipFree(deviceWrite);

	return 0;
}

//Fills a grid with a border of zeroes and inner portion with x * y
void fillGrid(int* grid) {
	//Fills in the leftmost and rightmost columns
	for (int y = 0; y < THREAD_DIM; y++) {
		grid[THREAD_DIM * y] = grid[THREAD_DIM * y + THREAD_DIM - 1] = 0;
	}

	//Fills in the top and bottom rows
	for (int x = 0; x < THREAD_DIM; x++) {
		grid[x] = grid[THREAD_DIM * (THREAD_DIM - 1) + x] = 0;
	}

	//Fills in each inner cell in the grid with a 1
	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			grid[THREAD_DIM * y + x] = 1;
		}
	}
}

//Performs a parallelized stencil computation
__global__ void computeGrid(int* read, int* write) {
	//Retrieve the thread's position in the grid
	int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int y = blockDim.y * blockIdx.y + threadIdx.y + 1;

	//Writes the sum of the neighbors to the cell
	write[THREAD_DIM * y + x] = read[THREAD_DIM * (y - 1) + x] + read[THREAD_DIM * (y + 1) + x] + read[THREAD_DIM * y + x - 1] + read[THREAD_DIM * y + x + 1];
}

//Swaps the pointers of two grids
void swapGrids(int** read, int** write) {
	int* temp = *read;
	*read = *write;
	*write = temp;
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid, int skip, const char* name) {
	printf("<<< %s >>>\n\n", name);

	for (int y = skip; y < THREAD_DIM - skip; y++) {
		for (int x = skip; x < THREAD_DIM - skip; x++) {
			printf("%-15d", grid[THREAD_DIM * y + x]);
		}

		printf("\n");
	}
}