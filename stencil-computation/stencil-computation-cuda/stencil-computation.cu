#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"


#define THREAD_DIM 12
#define GRID_DIM 5
#define BLOCK_DIM ((THREAD_DIM - 2) / GRID_DIM)
#define MEM_SIZE (sizeof(int) * THREAD_DIM * THREAD_DIM)
#define TIME_STEPS 1000000

void fillGrid(int* grid);
__global__ void computeGrid(int* read, int* write);
void swapGrids(int** read, int** write);
void printGrid(int* grid, int skip, const char* name);

int main() {
	if ((THREAD_DIM - 2) / GRID_DIM != 0) {
		printf("Error: bad grid/thread dimensions.\n");
		return 1;
	}

	//Allocates space on the host for the grids in pinned memory
	//Using pinned memory is much faster, only being limited by the speed of the PCI-E bus
	int* hostRead;
	int* hostWrite;
	hipHostMalloc((void**) &hostRead, MEM_SIZE, hipHostMallocDefault);
	hipHostMalloc((void**) &hostWrite, MEM_SIZE, hipHostMallocDefault);

	//Fills in the read grid serially
	fillGrid(hostRead);

	//Allocates space on the device for the grids, and copies over the input grid
	int* deviceRead;
	int* deviceWrite;
	hipMalloc(&deviceRead, MEM_SIZE);
	hipMalloc(&deviceWrite, MEM_SIZE);
	hipMemcpy(deviceRead, hostRead, MEM_SIZE, hipMemcpyHostToDevice);

	//Calls the computeGrid kernel TIME_STEPS times, swapping input and output each time
	dim3 gridDimension(GRID_DIM, GRID_DIM);
	dim3 blockDimension(BLOCK_DIM, BLOCK_DIM);
	for (int i = 0; i < TIME_STEPS; i++) {
		computeGrid<<<gridDimension, blockDimension>>>(deviceRead, deviceWrite);
		hipDeviceSynchronize();
		swapGrids(&deviceRead, &deviceWrite);
	}

	//Copies over the result (now in deviceRead) from device to host
	hipMemcpy(hostWrite, deviceRead, MEM_SIZE, hipMemcpyDeviceToHost);
	printGrid(hostWrite, 1, "Result");

	//Frees both the host and device grids
	hipHostFree(hostRead);
	hipHostFree(hostWrite);
	hipFree(deviceRead);
	hipFree(deviceWrite);

	return 0;
}

//Fills a grid with a border of zeroes and inner portion with x * y
void fillGrid(int* grid) {
	//Fills in the leftmost and rightmost columns
	for (int y = 0; y < THREAD_DIM; y++) {
		grid[THREAD_DIM * y] = grid[THREAD_DIM * y + THREAD_DIM - 1] = 0;
	}

	//Fills in the top and bottom rows
	for (int x = 0; x < THREAD_DIM; x++) {
		grid[x] = grid[THREAD_DIM * (THREAD_DIM - 1) + x] = 0;
	}

	//Fills in each inner cell in the grid with a 1
	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			grid[THREAD_DIM * y + x] = 1;
		}
	}
}

//Performs a parallelized stencil computation
__global__ void computeGrid(int* read, int* write) {
	//Retrieve the thread's position in the grid
	int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
	int y = blockDim.y * blockIdx.y + threadIdx.y + 1;

	//Writes the sum of the neighbors to the cell
	write[THREAD_DIM * y + x] = read[THREAD_DIM * (y - 1) + x] + read[THREAD_DIM * (y + 1) + x] + read[THREAD_DIM * y + x - 1] + read[THREAD_DIM * y + x + 1];
}

//Swaps the pointers of two grids
void swapGrids(int** read, int** write) {
	int* temp = *read;
	*read = *write;
	*write = temp;
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid, int skip, const char* name) {
	printf("<<< %s >>>\n\n", name);

	for (int y = skip; y < THREAD_DIM - skip; y++) {
		for (int x = skip; x < THREAD_DIM - skip; x++) {
			printf("%-15d", grid[THREAD_DIM * y + x]);
		}

		printf("\n");
	}
}

/*
#include <stdio.h>
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <math.h>

constexpr auto THREAD_DIM = 50; 				//Square dimensions of the ENTIRE grid, including inner grid, whose dimension is 2 less
constexpr auto GRID_DIM = 10;					//Square dimensions of the grid in blocks
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)		//Square dimensions of a single block, which only works if THREAD_DIM % GRID_DIM == 0

//Prototypes
__global__ void fillGrid(int* grid);
__global__ void computeGrid(int* grid, int* result);
void printGrid(int* grid, const char* name);

static int createOutput = 0;

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocate memory for the grid (1D array) and the result grid
	int* grid;
	int* result;
	cudaMallocManaged(&grid, THREAD_DIM * THREAD_DIM * sizeof(int));
	cudaMallocManaged(&result, THREAD_DIM * THREAD_DIM * sizeof(int));
	if (grid == NULL || result == NULL) {
		printf("Error: memory allocation failed.\n");
		return 1;
	}

	//Sets up the CUDA events to handle timing
	cudaEvent_t start;
	cudaEvent_t stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);
	float fillTime = 0;
	float computeTime = 0;

	//Fill in the grid, keeping time
	cudaEventRecord(start);
	fillGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid);
	cudaDeviceSynchronize();
	cudaEventRecord(stop);
	cudaEventSynchronize(stop);
	cudaEventElapsedTime(&fillTime, start, stop);

	//Compute the grid, keeping time
	cudaEventRecord(start);
	computeGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid, result);
	cudaDeviceSynchronize();
	cudaEventRecord(stop);
	cudaEventSynchronize(stop);
	cudaEventElapsedTime(&computeTime, start, stop);

	//Destroys the timing events
	cudaEventDestroy(start);
	cudaEventDestroy(stop);

	//Prints the elapsed time for both operations
	printf("Fill time: %.0fus\n", roundf(fillTime * 1000));
	printf("Compute time: %.0fus\n", roundf(computeTime * 1000));

	//If createOutput is 1, the grids will be printed out
	if (createOutput) {
		//Print out state of the filled and result grids
		printGrid(grid, "Filled");
		printf("\n");
		printGrid(result, "Result");
	}

	//Frees the memory used by the grids
	cudaFree(grid);
	cudaFree(result);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
__global__ void fillGrid(int* grid) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If the grid point is along the edges, fill in 0
	//Otherwise, fill in with value dependent on x and y values
	*(grid + THREAD_DIM * y + x) = x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1 ? 0 : x * y;
}

//Computes the result grid by adding all neighbors
__global__ void computeGrid(int* grid, int* result) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If along the edges of the grid, don't add neighbors
	if (x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1) {
		return;
	}

	//Put the sum of the neighbors in the result
	*(result + THREAD_DIM * y + x) = *(grid + THREAD_DIM * (y - 1) + x) + *(grid + THREAD_DIM * (y + 1) + x) + *(grid + THREAD_DIM * y + x - 1) + *(grid + THREAD_DIM * y + x + 1);
}
*/