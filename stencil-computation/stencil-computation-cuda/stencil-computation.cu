#include <stdio.h>
#include "hip/hip_runtime.h"

#include <math.h>

constexpr auto THREAD_DIM = 18000; 				//Square dimensions of the ENTIRE grid, including inner grid, whose dimension is 2 less
constexpr auto GRID_DIM = 10;					//Square dimensions of the grid in blocks
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)		//Square dimensions of a single block, which only works if THREAD_DIM % GRID_DIM == 0

//Prototypes
__global__ void fillGrid(int* grid);
__global__ void computeGrid(int* grid, int* result);
void printGrid(int* grid, const char* name);

static int createOutput = 0;

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocate memory for the grid (1D array) and the result grid
	int* grid;
	int* result;
	hipMallocManaged(&grid, THREAD_DIM * THREAD_DIM * sizeof(int));
	hipMallocManaged(&result, THREAD_DIM * THREAD_DIM * sizeof(int));
	if (grid == NULL || result == NULL) {
		printf("Error: memory allocation failed.\n");
		return 1;
	}

	//Sets up the CUDA events to handle timing
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float fillTime = 0;
	float computeTime = 0;

	//Fill in the grid, keeping time
	hipEventRecord(start);
	fillGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&fillTime, start, stop);

	//Compute the grid, keeping time
	hipEventRecord(start);
	computeGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid, result);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computeTime, start, stop);

	//Destroys the timing events
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//Prints the elapsed time for both operations
	printf("Fill time: %.0fus\n", roundf(fillTime * 1000));
	printf("Compute time: %.0fus\n", roundf(computeTime * 1000));

	//If createOutput is 1, the grids will be printed out
	if (createOutput) {
		//Print out state of the filled and result grids
		printGrid(grid, "Filled");
		printf("\n");
		printGrid(result, "Result");
	}

	//Frees the memory used by the grids
	hipFree(grid);
	hipFree(result);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
__global__ void fillGrid(int* grid) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If the grid point is along the edges, fill in 0
	//Otherwise, fill in with value dependent on x and y values
	*(grid + THREAD_DIM * y + x) = x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1 ? 0 : x * y;
}

//Computes the result grid by adding all neighbors
__global__ void computeGrid(int* grid, int* result) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If along the edges of the grid, don't add neighbors
	if (x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1) {
		return;
	}

	//Put the sum of the neighbors in the result
	*(result + THREAD_DIM * y + x) = *(grid + THREAD_DIM * (y - 1) + x) + *(grid + THREAD_DIM * (y + 1) + x) + *(grid + THREAD_DIM * y + x - 1) + *(grid + THREAD_DIM * y + x + 1);
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid, const char* name) {
	printf("<<< %s >>>\n\n", name);

	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			printf("%-10d", *(grid + THREAD_DIM * y + x));
		}

		printf("\n");
	}
}