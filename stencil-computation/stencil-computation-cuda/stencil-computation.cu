#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#define THREAD_DIM 20
#define GRID_DIM 4
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)
#define MEM_SIZE (sizeof(int) * THREAD_DIM * THREAD_DIM)
#define TIME_STEPS 100

void fillGrid(int* grid);
__global__ void computeGrid(int* read, int* write);
void swapGrids(int** read, int** write);

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocates space on the host for the grids
	int* hostRead = (int*)malloc(MEM_SIZE);
	int* hostWrite = (int*)malloc(MEM_SIZE);

	//Fills in the read grid serially
	fillGrid(hostRead);

	//Allocates space on the device for the grids, and copies over the input grid
	int* deviceRead;
	int* deviceWrite;
	hipMalloc(&deviceRead, MEM_SIZE);
	hipMalloc(&deviceWrite, MEM_SIZE);
	hipMemcpy(deviceRead, hostRead, MEM_SIZE, hipMemcpyHostToDevice);

	//Calls the computeGrid kernel TIME_STEPS times, swapping input and output each time
	dim3 gridDimension(GRID_DIM, GRID_DIM);
	dim3 blockDimension(BLOCK_DIM, BLOCK_DIM);
	for (int i = 0; i < TIME_STEPS; i++) {
		computeGrid << <gridDimension, blockDimension >> > (deviceRead, deviceWrite);
		swapGrids(&deviceRead, &deviceWrite);
	}

	//Copies over the result (now in deviceRead) from device to host
	hipMemcpy(hostWrite, deviceRead, MEM_SIZE, hipMemcpyDeviceToHost);

	//Frees both the host and device grids
	free(hostRead);
	free(hostWrite);
	hipFree(deviceRead);
	hipFree(deviceWrite);

	return 0;
}

//Fills a grid with a border of zeroes and inner portion with x * y
void fillGrid(int* grid) {
	//Fills in the leftmost and rightmost columns
	for (int r = 0; r < THREAD_DIM; r++) {
		grid[THREAD_DIM * r] = grid[THREAD_DIM * r + THREAD_DIM - 1] = 0;
	}

	//Fills in the top and bottom rows
	for (int c = 0; c < THREAD_DIM; c++) {
		grid[c] = grid[THREAD_DIM * (THREAD_DIM - 1) + c] = 0;
	}

	//Fills in each inner cell in the grid with the product of its x and y position
	for (int r = 1; r < THREAD_DIM - 1; r++) {
		for (int c = 1; c < THREAD_DIM - 1; c++) {
			grid[THREAD_DIM * r + c] = r * c;
		}
	}
}

//Performs a parallelized stencil computation
__global__ void computeGrid(int* read, int* write) {
	//Retrieve the thread's position in the grid
	int r = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.x * blockIdx.x + threadIdx.x;

	//If along the edges of the grid, do nothing
	if (r == 0 || c == 0 || r == THREAD_DIM - 1 || c == THREAD_DIM - 1) {
		return;
	}

	write[THREAD_DIM * r + c] = read[THREAD_DIM * (r - 1) + c] +
		read[THREAD_DIM * (r + 1) + c] +
		read[THREAD_DIM * r + c - 1] +
		read[THREAD_DIM * r + c + 1];
}

//Swaps the pointers of two grids
void swapGrids(int** read, int** write) {
	int* temp = *read;
	*read = *write;
	*write = temp;
}

/*
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

constexpr auto THREAD_DIM = 50; 				//Square dimensions of the ENTIRE grid, including inner grid, whose dimension is 2 less
constexpr auto GRID_DIM = 10;					//Square dimensions of the grid in blocks
#define BLOCK_DIM (THREAD_DIM / GRID_DIM)		//Square dimensions of a single block, which only works if THREAD_DIM % GRID_DIM == 0

//Prototypes
__global__ void fillGrid(int* grid);
__global__ void computeGrid(int* grid, int* result);
void printGrid(int* grid, const char* name);

static int createOutput = 0;

int main() {
	//Check for errors in grid size definitions
	if (THREAD_DIM % GRID_DIM != 0) {
		printf("Error: bad grid size definitions.\n");
		return 1;
	}

	//Allocate memory for the grid (1D array) and the result grid
	int* grid;
	int* result;
	hipMallocManaged(&grid, THREAD_DIM * THREAD_DIM * sizeof(int));
	hipMallocManaged(&result, THREAD_DIM * THREAD_DIM * sizeof(int));
	if (grid == NULL || result == NULL) {
		printf("Error: memory allocation failed.\n");
		return 1;
	}

	//Sets up the CUDA events to handle timing
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float fillTime = 0;
	float computeTime = 0;

	//Fill in the grid, keeping time
	hipEventRecord(start);
	fillGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&fillTime, start, stop);

	//Compute the grid, keeping time
	hipEventRecord(start);
	computeGrid<<<dim3(GRID_DIM, GRID_DIM), dim3(BLOCK_DIM, BLOCK_DIM)>>>(grid, result);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computeTime, start, stop);

	//Destroys the timing events
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//Prints the elapsed time for both operations
	printf("Fill time: %.0fus\n", roundf(fillTime * 1000));
	printf("Compute time: %.0fus\n", roundf(computeTime * 1000));

	//If createOutput is 1, the grids will be printed out
	if (createOutput) {
		//Print out state of the filled and result grids
		printGrid(grid, "Filled");
		printf("\n");
		printGrid(result, "Result");
	}

	//Frees the memory used by the grids
	hipFree(grid);
	hipFree(result);

	return 0;
}

//Fills in the grid based on a thread's position in the grid
__global__ void fillGrid(int* grid) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If the grid point is along the edges, fill in 0
	//Otherwise, fill in with value dependent on x and y values
	*(grid + THREAD_DIM * y + x) = x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1 ? 0 : x * y;
}

//Computes the result grid by adding all neighbors
__global__ void computeGrid(int* grid, int* result) {
	//Retrieve the thread's position in the grid
	//x is the column, y is the row
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;

	//If along the edges of the grid, don't add neighbors
	if (x == 0 || y == 0 || x == THREAD_DIM - 1 || y == THREAD_DIM - 1) {
		return;
	}

	//Put the sum of the neighbors in the result
	*(result + THREAD_DIM * y + x) = *(grid + THREAD_DIM * (y - 1) + x) + *(grid + THREAD_DIM * (y + 1) + x) + *(grid + THREAD_DIM * y + x - 1) + *(grid + THREAD_DIM * y + x + 1);
}

//Prints out the state of the internal grid (can also print entire grid)
void printGrid(int* grid, const char* name) {
	printf("<<< %s >>>\n\n", name);

	for (int y = 1; y < THREAD_DIM - 1; y++) {
		for (int x = 1; x < THREAD_DIM - 1; x++) {
			printf("%-10d", *(grid + THREAD_DIM * y + x));
		}

		printf("\n");
	}
}
*/